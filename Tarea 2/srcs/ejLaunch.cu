#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>

__global__ void hola(int n, float a, float *x) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    printf(" i %d blockDim.x %d gridDim.x %d\n", i, blockDim.x, gridDim.x);
    if (i < n) x[i] = a*x[i];
}

int main(void) {
    int N = 1 << 4;
    unsigned int threads = 2;
    float *hx, *dx;
    hx = (float*)malloc(N * sizeof(float));

    hipMalloc(&dx, N * sizeof(float));

    for (int i = 0; i < N; i++)
    {
        hx[i] = (float)(N -i);
    }

    hipMemcpy(dx, hx, N * sizeof(float), hipMemcpyHostToDevice);

    //unsigned int blocks = (N + 255) / threads;
    unsigned int blocks = ceil(N/threads);

    float a = 2.0f;

    void *args[] = { &N, &a, &dx };
    hipError_t res = hipLaunchKernel(reinterpret_cast<const void*>((void*)hola), dim3(blocks,1,1), dim3(threads,1,1), args, 0, NULL);
    if (res != hipSuccess) {
        printf ("error en kernel launch: %s \n", hipGetErrorString(res));
        return -1;
    }
    hipDeviceSynchronize();


    hipMemcpy(hx, dx, N * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++) {
	printf(" hx %f\n",hx[i]);
    }

    hipFree(dx);
    free(hx);

    return 0;
}

