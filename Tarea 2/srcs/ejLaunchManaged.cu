#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>

__global__ void hola(int n, float a, float *x) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) x[i] = a*x[i];
}

int main(void) {
    int N = 1 << 4;
    unsigned int threads = 2;
    float *hdx;

    hipMallocManaged(&hdx, N * sizeof(float));

    for (int i = 0; i < N; i++)
    {
        hdx[i] = (float)(N -i);
    }
    unsigned int blocks = ceil(N/threads);

    float a = 2.0f;

    void *args[] = { &N, &a, &hdx };
    hipError_t res = hipLaunchKernel(reinterpret_cast<const void*>((void*)hola), dim3(blocks,1,1), dim3(threads,1,1), args, 0, NULL);
    if (res != hipSuccess) {
        printf ("error en kernel launch: %s \n", hipGetErrorString(res));
        return -1;
    }
    hipDeviceSynchronize();


    for (int i = 0; i < N; i++) {
	printf(" hdx %f\n",hdx[i]);
    }

    hipFree(hdx);

    return 0;
}

