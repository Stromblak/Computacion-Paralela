#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>

using namespace cooperative_groups;

__device__ void holatg(thread_group g) {
    int tid = g.thread_rank();
    printf(" tid %d\n", tid);
}

__global__ void hola(int n, float a, float *x) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    grid_group g = this_grid();
    printf(" grid size %llu x %f\n", g.size(), x[i]);
    thread_group tg = this_thread_block();
    holatg(tg);

    if (i < n) x[i] = a*x[i];
}

int main(void) {
    int N = 1 << 4;
    unsigned int threads = 2;
    float *hx, *dx;
    hx = (float*)malloc(N * sizeof(float));

    hipMalloc(&dx, N * sizeof(float));

    for (int i = 0; i < N; i++)
    {
        hx[i] = (float)(N -i);
    }

    hipMemcpy(dx, hx, N * sizeof(float), hipMemcpyHostToDevice);

    //unsigned int blocks = (N + 255) / threads;
    unsigned int blocks = ceil(N/threads);

    float a = 2.0f;

    void *args[] = { &N, &a, &dx };
    hipError_t res = hipLaunchKernel((void*)hola, dim3(blocks,1,1), dim3(threads,1,1), args, 0, NULL);
    if (res != hipSuccess) {
        printf ("error en kernel launch: %s \n", hipGetErrorString(res));
        return -1;
    }

    hipMemcpy(hx, dx, N * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++) {
	printf(" hx %f\n",hx[i]);
    }

    hipFree(dx);
    free(hx);

    return 0;
}

