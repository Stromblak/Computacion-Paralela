#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>

__global__ void doble(float *x, int offset) {
    int i = offset + blockIdx.x * blockDim.x + threadIdx.x;
    x[i] = 2*x[i];
}

int main(void) {
    int N = 1 << 4;
    unsigned int threads = 2;
    float *hx, *dx;

    hipHostMalloc(&hx, N * sizeof(float), hipHostMallocDefault);
    hipMalloc(&dx, N * sizeof(float));

    for (int i = 0; i < N; i++)
    {
        hx[i] = (float)(N -i);
    }
    int nStreams = 2;
    int streamSize = N/nStreams;
    int streamSizeBytes = streamSize*sizeof(int);
    int gdstream = streamSize/threads;
    hipStream_t stream[nStreams];
    for (int i = 0; i < nStreams; i ++) {
        hipStreamCreate(&stream[i]);
    }
    printf(" N %d blocks %d streamSize %d gdstream %d\n", N, threads, streamSize, gdstream);
    for (int i = 0; i < nStreams; i ++) {
        int offset = i * streamSize;
        hipMemcpyAsync(&dx[offset], &hx[offset], streamSizeBytes, hipMemcpyHostToDevice, stream[i]);
        doble<<<gdstream, threads, 0, stream[i]>>>(dx, offset);
        hipMemcpyAsync(&hx[offset], &dx[offset], streamSizeBytes, hipMemcpyDeviceToHost, stream[i]);
    }
    hipDeviceSynchronize();

    for (int i = 0; i < N; i++) {
	printf(" hx %f\n",hx[i]);
    }

    hipFree(dx);
    hipHostFree(hx);
    for (int i = 0; i < nStreams; i ++) {
        hipStreamDestroy(stream[i]);
    }

    return 0;
}

